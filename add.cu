#include "hip/hip_runtime.h"
#include <iostream>

// kernel method
__global__ void add(int a, int b, int *c)
{
	*c = a+b;
}

int main(void)
{
	int c, *dev_c;

	//address that points to *dev_c
	//deref dev_c and points to gpu_mem
	hipMalloc ( (void **) &dev_c, sizeof(int) );

	add <<<1,1>>> (5,4,dev_c);

	hipMemcpy (&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	std::cout << c << endl;

	hipFree(dev_c);

	return 0;
}