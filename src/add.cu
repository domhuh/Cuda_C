
#include <hip/hip_runtime.h>
#include <iostream>

// kernel method
__global__ void add(int a, int b, int *c)
{
	*c = a+b;
}

int main(void)
{
	int c, *dev_c;

	//address that points to *dev_c
	//deref dev_c and points to gpu_mem
	hipMalloc ( (void **) &dev_c, sizeof(int) );

	add <<<1,1>>> (5,4,dev_c);

	hipMemcpy (&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);

	std::cout << c;


	return 0;
}