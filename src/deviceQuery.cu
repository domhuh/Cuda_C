#include "hip/hip_runtime.h"
#include <iostream>
/*
export PATH=/Developer/NVIDIA/CUDA-10.2/bin${PATH:+:${PATH}}
export DYLD_LIBRARY_PATH=/Developer/NVIDIA/CUDA-10.2/lib\${DYLD_LIBRARY_PATH:+:${DYLD_LIBRARY_PATH}}
*/
__global__ void kernel(){
	
}


int main(void){
	int count;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);
	
	for (int i=0;i<count;i++){
		hipGetDeviceProperties(&prop, i);
		std::cout<< prop.name<< std::endl;
		std::cout<< prop.major << prop.minor << std::endl;
		std::cout<< prop.clockRate << std::endl;
	}


	return 0;
}